
#include <hip/hip_runtime.h>
/**
 * KERNEL cuAdd() - Takes 2 input arrays of same size N and adds them into C.
 * Locations are found by computing the global index of each thread.
 * @return 
 */
__global__ void cuAdd(int *a,int *b,int *c, int N)
{
	// global index
    int offset = blockDim.x * blockIdx.x + threadIdx.x;
    if(offset < N)
    {
        c[offset] = a[offset] + b[offset];
    }
}
