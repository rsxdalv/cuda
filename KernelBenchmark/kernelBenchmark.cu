
#include <hip/hip_runtime.h>
// Kernel pointer typedef
typedef void (*MatrixMult)(float *a, float *b, float *c, int wA, int wB, int hA);

// 
struct cudaLaunchError_t {
    hipError_t error;
    const char * action;
};

float d_Benchmark_MM(
        MatrixMult kernel, // Kernel pointer
        const char *name, // Name of kernel for identification
        dim3 gridSize, dim3 blockSize, // common launch parameters for all kernels
        float * _a, float * _b, float * _c, int wA, int wB, int hA) // kernel arguments
{
    hipError_t error;
    hipEvent_t start, stop;
    
    float GEMM_ms = 0.f;
    
    try
    {
        error = hipEventCreate(&start);
        if (error != hipSuccess)
            throw (struct cudaLaunchError_t){error, "create start event"};

        error = hipEventCreate(&stop);
        if (error != hipSuccess)
            throw (struct cudaLaunchError_t){error, "create stop event"};
     
        // Record the start event
        error = hipEventRecord(start, NULL);
        if (error != hipSuccess)
            throw (struct cudaLaunchError_t){error, "record start event"};

        // Kernel invocation
        kernel<<<gridSize, blockSize >>>(_a, _b, _c, wA, wB, hA);

        // Record the stop event
        error = hipEventRecord(stop, NULL);
        if (error != hipSuccess)
            throw (struct cudaLaunchError_t){error, "record stop event"};

        // Wait for the stop event to complete
        error = hipEventSynchronize(stop);
        if (error != hipSuccess)
            throw (struct cudaLaunchError_t){error, "synchronize on the stop event"};
        
        error = hipEventElapsedTime(&GEMM_ms, start, stop);
        if (error != hipSuccess)
            throw (struct cudaLaunchError_t){error, "get time elapsed between events"};
    }
    catch(struct cudaLaunchError_t e)
    {
        fprintf(stderr, "Failed to %s (error code %s)!\n", e.action, hipGetErrorString(e.error));
        exit(EXIT_FAILURE);
    }
        
    int wC = wB;
    int hC = hA;
    
    // Calculate the number of FLOP
    const double FLOP_GEMM = 1.0 * wC * hC * wA;
    // Calculate the gigaflops per second
    double gigaFLOPS = (FLOP_GEMM * 1.0e-9f) / (GEMM_ms / 1000.f);
    
    // Print the results in a table
    fprintf(stderr, "Benchmark of %s results:\n"
            "%4.4f GFLOPS \t%4.4fms \t WorkgroupSize= %u threads/block\n",
                                name,
            gigaFLOPS,      GEMM_ms,    blockSize.x * blockSize.y);
    
    return GEMM_ms;
}
