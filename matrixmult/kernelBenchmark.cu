enum KernelCode {
    k_MM,
    k_MM_OPT
};

float d_Benchmark_MM(enum KernelCode kid,  // kernel specifier
        //hipError_t & error, hipEvent_t & start, hipEvent_t & stop,
        dim3 gridSize, dim3 blockSize, // common launch parameters for all kernels
        float * _a, float * _b, float * _c, int wA, int wB, int hA) // kernel arguments
{
    hipError_t error;
    hipEvent_t start, stop;
    
    error = hipEventCreate(&start);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to create start event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    error = hipEventCreate(&stop);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to create stop event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to record start event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    // kernel call
    switch(kid)
    {
        case k_MM:
            fprintf(stderr, "Benchmark of d_MM \n");
            d_MM<<< gridSize, blockSize >>>(_a, _b, _c, wA, wB, hA);
            break;
        case k_MM_OPT:
            fprintf(stderr, "Benchmark of d_MM_OPT \n");
            d_MM_OPT<<< gridSize, blockSize >>>(_a, _b, _c, wA, wB, hA);
            break;
        default:
            fprintf(stderr, "No Kernel Code!\n");
            return 0.f;
    }

    // Record the stop event
    error = hipEventRecord(stop, NULL);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to record stop event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    float GEMM_ms = 0.f;
    error = hipEventElapsedTime(&GEMM_ms, start, stop);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }
    
    int wC = wB;
    int hC = hA;
    
    // Calculate the number of FLOP
    const double FLOP_GEMM = 1.0 * wC * hC * wA;
    // Calculate the giga flops per second
    double gigaFLOPS = (FLOP_GEMM * 1.0e-9f) / (GEMM_ms / 1000.f);
    
    // Print the results in a table
    printf("Results:\n %4.4f GFLOPS \t%4.4fms \t WorkgroupSize= %u threads/block\n",
                    gigaFLOPS,
                    GEMM_ms,
                    blockSize.x * blockSize.y);
    
    return GEMM_ms;
}
