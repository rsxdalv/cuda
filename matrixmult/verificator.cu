#include "hip/hip_runtime.h"
int VerifyCalculation(float *c, float *hh_c, float threshold)
{
    int errors = 0;
    for( int k = 0; k < wB*hA; k++)
    {
        /* Make sure absolute difference is below a threshold */
        if(abs(c[k] - hh_c[k]) > threshold)
            errors++;
    }
    printf("Errors: %d\n", errors);
}