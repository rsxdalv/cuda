
#include <hip/hip_runtime.h>
#include <sys/time.h>

double microSeconds()
{
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, NULL);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}