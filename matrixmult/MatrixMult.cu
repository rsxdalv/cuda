
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

/**
 * KERNEL cuAdd() - Takes 2 input arrays of same size N and adds them into C.
 * Locations are found by computing the global index of each thread.
 * @return 
 */
__global__ void cuAdd(int *a,int *b,int *c, int N)
{
	// 1D global index
    int offset = blockDim.x * blockIdx.x + threadIdx.x;
    if(offset < N)
    {
        c[offset] = a[offset] + b[offset];
    }
}

/**
 * KERNL cuMult() - Takes two 2D matrices and multiplies them
 * @param a - 1st Matrix
 * @param b - 2nd Matrix
 * @param c - Result Matrix
 * @param wA - length of A and depth of B
 * @param wB - length of matrix B and C
 * @param hA - depth of matrix A and C
 */
__global__ void cuMult(int *a, int *b, int *c, int wA, int wB, int hA)
{
    // global index
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;  // col
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;   // row
    
    if(gidx < wB && gidy < hA)
    {
        int sum = 0;
        for(int k=0;k<wA;k++)
        {
            sum += a[gidy*hA + k] * b[k*wB +gidx];
        }
        c[gidy * wB + gidx] = sum;
    }
}

/**
 * KERNL cuMultOpti() - Takes two 2D matrices and multiplies them optimally
 * @param a - 1st Matrix
 * @param b - 2nd Matrix
 * @param c - Result Matrix
 * @param wA - length of A and depth of B
 * @param wB - length of matrix B and C
 * @param hA - depth of matrix A and C
 */
__global__ void cuMultOpti(
        int *a,
        int *b,
        int *c,
        int wA,
        int wB,
        int hA)
{
#define blockTile 16
    /* Blocksize is 16x16 */
    /* Allocate shared memory */
    __shared__ int aBlock[blockTile][blockTile];
    __shared__ int bBlock[blockTile][blockTile];
    
    /* Calculate global index X, Y*/
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;  // column
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;   // row
    
    /* Assign shared memory and sync  */
    /* Warning, wA*gidy may be out of bounds */
    aBlock[threadIdx.x][threadIdx.y] = a[gidy*wA + threadIdx.x];
    bBlock[threadIdx.x][threadIdx.y] = b[threadIdx.y*wB + gidx];
    
    __syncthreads();
    
    /* Check if global IDs are within limits */
    if(gidx < wB && gidy < hA)
    {
        int sum = 0;
        for(int k=0; k<wA; k++)
        {
            sum += aBlock[threadIdx.y][k] * bBlock[k][threadIdx.x];
        }
        // c [gidy][gidx]
        c[gidy * wB + gidx] = sum;
    }
}

/**
 * HOST h_MatrixMult_Naive() - Takes two 2D matrices and multiplies them naively
 * @param a - 1st Matrix
 * @param b - 2nd Matrix
 * @param c - Result Matrix
 * @param wA - length of A and depth of B
 * @param wB - length of matrix B and C
 * @param hA - depth of matrix A and C
 */
void h_MatrixMult_Naive(
        int *a,
        int *b,
        int *c,
        int wA,
        int wB,
        int hA)
{
    // Iterate through all rows of a
    for(int i=0; i<hA; i++)
    {
        // Iterate through all columns of b
        for(int j=0; j<wB; j++)
        {
            // Calculate all of c[i][j] products
            int sum = 0;
            for(int k=0; k<wA; k++)
            {
                sum += a[i*hA + k] * b[k*wB + j];
            }
            // Index - row i of column j with column width of wB
            c[i * wB + j] = sum;
        }
    }
}

/**
 * ENTRY main() - Tests <<<>>>cuMult() kernel: Initializes memory and data on
 * the host, then memory on the device. Copies the data from host to device,
 * executes kernel with memory device pointers, copies result back to host,
 * displays results for error checking and frees allocated memory.
 * @return 
 */
int main(int argc, char ** argv)
{
    /**
     *  Neutral - both for host and device */
    
    // width A
    int wA = 320;
    // height A
    int hA = 640;
    
    // width B
    int wB = 320;
    // height B
    int hB = 320;
    
    int wC = wB;
    int hC = hA;

    size_t size_a = sizeof(int) * wA * hA;
    size_t size_b = sizeof(int) * wB * hB;
    size_t size_c = sizeof(int) * wC * hC;
	
    
    // host 
    int *a, *b, *c;//, *hh_c;
    a = (int *) malloc(size_a);
    b = (int *) malloc(size_b);
    c = (int *) malloc(size_c);
    /* Host test memory */
    //hh_c = (int *) malloc(size_c);
    
    //assert(hh_c != NULL);
    
    /**
     *  Device specific */
    
    // device
    int *_a, *_b, *_c;
    hipMalloc( (void **) &_a, size_a );
    hipMalloc( (void **) &_b, size_b );
    hipMalloc( (void **) &_c, size_c );

    /**
     Neutral */
    // initialize A
    for(int i=0; i < hA * wA; i++)
    {
        a[i] = 1;
    }
    
    // initialize B
    for(int i=0; i < hB * wB; i++)
    {
        b[i] = 2;
    }
    
    /**
     Device*/
    
    // copy data to GPU
    hipMemcpy(_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(_b, b, size_b, hipMemcpyHostToDevice);

    // x : col , y: row
    dim3 blockSize(16,16); 
    // (N.x + blockSize.x - 1)/blockSize.x, (N.y + blockSize.y -1)/blockSize.y)
    dim3 gridSize((wC+15)/16, (hC+15)/16);
        
    // kernel execution
    cuMult<<< gridSize, blockSize >>>(_a, _b, _c, wA, wB, hA);
    //cuMultOpti<<< gridSize, blockSize >>>(_a, _b, _c, wA, wB, hA);

    // copy data back to CPU
    hipMemcpy(c, _c, size_c, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // compare with cpu results
    
    /**
     Host*/
    
    //h_MatrixMult_Naive(a, b, hh_c, wA, wB, hA);
    
    // Check first and last memory location
    printf("Start: %d. Finish: %d.\n",c[2], c[wC * hC - 1]);
    
    /* Check */
    // Naive check
    int k = 0;
    while(c[k] == c[k+1])
        k++;
    printf("EQ Test: Breakpoint @ %d\n",k);
    // Device - Host check
    k = 0;
    //while(c[k] == hh_c[k])
        k++;
    printf("H2D Test: Breakpoint @ %d\n",k);

    // release resources
    hipFree(_a);
    hipFree(_b);
    hipFree(_c);

    free(a);
    free(b);
    free(c);
//    free(hh_c);

    return 0;
}
