#include "hip/hip_runtime.h"
/**
 * TODO: 
 *      Description including all the framework (Launch pad, Testing suite, Glue, Theory)
 *      Add function-lists to includes
 *      Create register pressure aware kernels
 *      Improve test suite code and output
 */
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>

// double microseconds();
#include "utils.cu"

#include "kernels.cu"

#include "hostKernels.cu"

/**
 * ENTRY main() - Tests <<<>>>cuMult() kernel: Initializes memory and data on
 * the host, then memory on the device. Copies the data from host to device,
 * executes kernel with memory device pointers, copies result back to host,
 * displays results for error checking and frees allocated memory.
 * @return 
 */
int main(int argc, char ** argv)
{
    // width A
    int wA = 512;
    // height A
    int hA = 512;
    
    // width B
    int wB = 512;
    // height B
    int hB = wA;
    
    // value A
    float aValue = 1.0;
    // value B
    float bValue = 2.0;
    
    /* Fetch the test parameters */
    if(argc < 6)
    {
        printf("Using default parameters: 320 640 320 1 2\n");
    }
    else
    {
        wA = atoi(argv[1]);
        hA = atoi(argv[2]);
        wB = atoi(argv[3]);
        hB = wA;
        aValue = atoi(argv[4]);
        bValue = atoi(argv[5]);
    }
    /**
     *  Neutral - both for host and device */
    
    int wC = wB;
    int hC = hA;

    size_t size_a = sizeof(float) * wA * hA;
    size_t size_b = sizeof(float) * wB * hB;
    size_t size_c = sizeof(float) * wC * hC;
	
    
    /* Host memory initialization */
    float *a, *b, *c, *hh_c;
    a = (float *) malloc(size_a);
    b = (float *) malloc(size_b);
    c = (float *) malloc(size_c);
    /* Host testing memory */
    hh_c = (float *) malloc(size_c);
    
    assert(hh_c != NULL);
    
    /* Device Memory Initialization */
    float *_a, *_b, *_c;
    hipMalloc( (void **) &_a, size_a );
    hipMalloc( (void **) &_b, size_b );
    hipMalloc( (void **) &_c, size_c );

    /* Input initialization */
    for(int i=0; i < hA * wA; i++)
    {
        a[i] = aValue;
    }
    
    for(int i=0; i < hB * wB; i++)
    {
        b[i] = bValue;
    }
    
    /* 
     * Device 
     */
    
    // copy data to GPU
    hipMemcpy(_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(_b, b, size_b, hipMemcpyHostToDevice);

    /* TODO: Write _macro_ for the rounded off gridSize calculation */
    // x : col , y: row
    dim3 blockSize(16,16);
    // (N.x + blockSize.x - 1)/blockSize.x, (N.y + blockSize.y -1)/blockSize.y)
    dim3 gridSize((wC+15)/16, (hC+15)/16);
        
    hipError_t error;

    hipEvent_t start, stop;

    ///////////////////////////////////////////////////
    // OPTIMIZED (What is?)
    
    error = hipEventCreate(&start);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to create start event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to create stop event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to record start event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }


    // kernel execution
    cuMult<<< gridSize, blockSize >>>(_a, _b, _c, wA, wB, hA);

    // Record the stop event
    error = hipEventRecord(stop, NULL);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to record stop event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    float sgemm_msec = 0.f;
    error = hipEventElapsedTime(&sgemm_msec, start, stop);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }



    /* Comments about GEMM of the benchmark toolkit */
    // C := alpha*op( A )*op( B ) + beta*C
    // GEMM performs 4 floating point operations for one data output
    //double flops_sgemm = 4.0 * (double) NI * (double) NJ * (double) NK;

    //double gigaFlops = (flops_sgemm * 1.0e-9f) / (sgemm_msec / 1000.f);

    printf("%.4f\t", sgemm_msec);
    printf("N_Time: %.3f\n, WorkgroupSize= %u threads/block\n",
                    //gigaFlops,
                    sgemm_msec,
                    //flops_sgemm,
                    blockSize.x * blockSize.y);
    
    
    // copy data back to CPU
    hipMemcpy(c, _c, size_c, hipMemcpyDeviceToHost);
    
    /////////////////////////////////////////////////
    // OPTIMIZED
    
    error = hipEventCreate(&start);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to create start event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to create stop event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to record start event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }


    // kernel execution
    cuMultOpti<<< gridSize, blockSize >>>(_a, _b, _c, wA, wB, hA);

    // Record the stop event
    error = hipEventRecord(stop, NULL);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to record stop event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }

    sgemm_msec = 0.f;
    error = hipEventElapsedTime(&sgemm_msec, start, stop);
    if (error != hipSuccess)
    {
            fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", 
            hipGetErrorString(error));
            exit(EXIT_FAILURE);
    }
    
    printf("O_Time: %.3f\nWorkgroupSize= %u threads/block\n",
                    //gigaFlops,
                    sgemm_msec,
                    //flops_sgemm,
                    blockSize.x * blockSize.y);
    printf("%.4f\t", sgemm_msec);
    
    
    
    // copy data back to CPU
    hipMemcpy(c, _c, size_c, hipMemcpyDeviceToHost);
    
    //////////////////////////////////////////////////
    // HOST
    
    // compare with cpu results
    /**
     Host*/
    double h_start, h_end;
    h_start = microSeconds();
    h_MatrixMult_Naive(a, b, hh_c, wA, wB, hA);
    h_end = microSeconds();
    
    printf("%4.4f\t", (h_end - h_start) * 1000);
    
    /* Check */
//    // Naive check
//    int k = 0;
//    while(c[k] == c[k+1])
//        k++;
//    printf("EQ Test: Breakpoint @ %d\n",k);
    // Check first and last memory location
    //printf("Start: %d. Finish: %d.\n",c[2], c[wC * hC - 1]);
    
    /* TODO: Create test function */
    int fail = 0;
    for( int k = 0; k< wB*hA; k++)
    {
        if(abs(c[k] - hh_c[k]) > 1e-5)
            fail++;
    }
    printf("\nWorkgroup: %d Data: %d Failures: %d\n", blockSize.x*blockSize.y, wC, fail);

    // release resources
    hipFree(_a);
    hipFree(_b);
    hipFree(_c);

    free(a);
    free(b);
    free(c);
    free(hh_c);

    return 0;
}
