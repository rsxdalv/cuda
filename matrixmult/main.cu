/**
 * Project details:
 *      Host: kepler1 (Ubuntu 14.04 LTS)
 *      Hardware: GPUs k40c k20c
 *      IDE: Netbeans 8.0.2
 *      Goal: Benchmark highly optimized matrix multiplication with parallelization
 * 
 * TODO: 
 *      Create register pressure aware kernels
 */
// Some standard libraries are included as per CMake configuration.
// TODO: Add the list of them /usr/local/cuda/lib64

// printf() - Text output via fprintf()
#include <stdio.h>
// getopt() - Command line argument parsing
#include <unistd.h>

// d_MM(), d_MM_OPT()
#include "kernels.cu"
// h_MM()
#include "hostKernels.cu"

// d_Benchmark()
#include "kernelBenchmark.cu"
// h_Benchmark()
#include "hostBenchmark.cu"

// VerifyCalculation()
#include "verificator.cu"

/**
 * Tests matrix multiplication on 2 kernels and 1 host algorithm, by setting up
 * the variables, allocating and initializing the memory, measuring the time,
 * showing the results, and cleaning up.
 */
int main(int argc, char ** argv)
{
    // width 'a', height 'h', width 'b'
    int     wA = 512,
            hA = 512,
            wB = 512;
    
    // value 'x', value 'y'
    float   aValue = 1.0,
            bValue = 2.0;
    
    // Disable getopt() error handling
    opterr = 0;
    int getopt_state = 0;
    while ((getopt_state = getopt (argc, argv, "a:h:b:x:y:")) != -1)
        switch (getopt_state)
        {
            case 'a':
                wA = atoi(optarg);
                break;
            case 'h':
                hA = atoi(optarg);
                break;
            case 'b':
                wB = atoi(optarg);
                break;
            case 'x':
                aValue = atoi(optarg);
                break;
            case 'y':
                bValue = atoi(optarg);
                break;
            case '?':
                fprintf(stderr, "Invalid Option or Missing argument for: -%c\n", optopt);
                break;
            default:
                fprintf(stderr, "GetOpt failure or uncaught option!\n");
                break;
        }
        
    printf("wA\thA\twB\ta\tb\n%d\t%d\t%d\t%1.2f\t%1.2f\n", wA, hA, wB, aValue, bValue);
    
    /**
     *  Neutral - both for host and device */
    
    int hB = wA;
    
    int wC = wB;
    int hC = hA;

    size_t size_a = sizeof(float) * wA * hA;
    size_t size_b = sizeof(float) * wB * hB;
    size_t size_c = sizeof(float) * wC * hC;
	
    
    /* Host memory initialization */
    float *a, *b, *c, *hh_c;
    a = (float *) malloc(size_a);
    b = (float *) malloc(size_b);
    c = (float *) malloc(size_c);
    /* Host testing memory */
    hh_c = (float *) malloc(size_c);
    
    //assert(hh_c != NULL);
    
    /* Device Memory Initialization */
    float *_a, *_b, *_c;
    hipMalloc( (void **) &_a, size_a );
    hipMalloc( (void **) &_b, size_b );
    hipMalloc( (void **) &_c, size_c );

    /* Input initialization */
    for(int i = 0; i < hA * wA; i++)
        a[i] = aValue;
    
    for(int i = 0; i < hB * wB; i++)
        b[i] = bValue;
    
    /* 
     * Device Specific Routine
     */
    
    // copy initialized data to GPU
    hipMemcpy(_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(_b, b, size_b, hipMemcpyHostToDevice);

    // x : columns , y: rows
#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16
    
    // Shorthand for int rounding
#define gridRound(width, blocksize) (width + blocksize - 1)/blocksize
    // (N.x + blockSize.x - 1)/blockSize.x, (N.y + blockSize.y -1)/blockSize.y)
    
    dim3 blockSize(BLOCKSIZE_X, BLOCKSIZE_Y);
    
    dim3 gridSize(gridRound(wC, BLOCKSIZE_X),
            gridRound(hC, BLOCKSIZE_Y));
    
    // Benchmark Matrix Multiplication Naive kernel
    d_Benchmark_MM(k_MM,
            //error, start, stop,
            gridSize, blockSize,
            _a, _b, _c, wA, wB, hA);
    
    // Benchmark Matrix Multiplication Optimized kernel
    d_Benchmark_MM(k_MM_OPT,
            //error, start, stop,
            gridSize, blockSize,
            _a, _b, _c, wA, wB, hA);

    // copy data back to CPU
    hipMemcpy(c, _c, size_c, hipMemcpyDeviceToHost);
    
    //////////////////////////////////////////////////
    h_Benchmark(a, b, hh_c, wA, wB, hA);

    VerifyCalculation(c, hh_c, wB*hA, 1e-5);
    
    // release resources
    hipFree(_a);
    hipFree(_b);
    hipFree(_c);

    free(a);
    free(b);
    free(c);
    free(hh_c);

    return 0;
}
