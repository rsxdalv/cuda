
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK   1024

__global__
void cuAdd(int *a,int *b,int *c, int N)
{
    int offset = blockDim.x * blockIdx.x + threadIdx.x;
    if(offset < N)
    {
        c[offset] = a[offset] + b[offset];
    }
}

#define N (1<<20)

int main()
{
    int *a, *b, *c,
        *_a, *_b, *_c;

    const int length = N * sizeof( int );

    hipMalloc( (void **) &_a, length );
    hipMalloc( (void **) &_b, length );
    hipMalloc( (void **) &_c, length );

    a = (int *) malloc(length);
    b = (int *) malloc(length);
    c = (int *) malloc(length);

    for(int i=0; i < N; i++)
    {
        a[i]=b[i]=i;
        c[i]=-1;
    }

    hipMemcpy(_a, a, length, hipMemcpyHostToDevice);
    hipMemcpy(_b, b, length, hipMemcpyHostToDevice);

    //int blocks = length/THREADS_PER_BLOCK;
    cuAdd<<<128, THREADS_PER_BLOCK>>>(_a,_b,_c, length);

    hipMemcpy(c, _c, length, hipMemcpyDeviceToHost);

    printf("Start: %d. Finish: %d.\n",c[0], c[N-1]);

    hipFree(_a);
    hipFree(_b);
    hipFree(_c);

    free(a);
    free(b);
    free(c);


    return 0;
}
