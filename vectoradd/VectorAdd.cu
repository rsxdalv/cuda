
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuAdd(int *a,int *b,int *c, int N)
{
	// global index
    int offset = blockDim.x * blockIdx.x + threadIdx.x;
    if(offset < N)
    {
        c[offset] = a[offset] + b[offset];
    }
}

#define N (1<<20)

int main()
{
    const int length = N * sizeof( int );

	// host 
    int *a, *b, *c;
    a = (int *) malloc(length);
    b = (int *) malloc(length);
    c = (int *) malloc(length);

	// device
	int *_a, *_b, *_c;
    hipMalloc( (void **) &_a, length );
    hipMalloc( (void **) &_b, length );
    hipMalloc( (void **) &_c, length );

	// initialize data on the cpu
    for(int i=0; i < N; i++)
    {
        a[i]=1;
		b[i]=2;
    }

	// copy data to gpu
    hipMemcpy(_a, a, length, hipMemcpyHostToDevice);
    hipMemcpy(_b, b, length, hipMemcpyHostToDevice);

	size_t blockSize = 1024; 
	size_t gridSize  = (N + blockSize - 1)/blockSize;

	// kernel execution
    cuAdd<<< gridSize, blockSize>>>(_a, _b, _c, length);

	// copy data back to cpu
    hipMemcpy(c, _c, length, hipMemcpyDeviceToHost);

    printf("Start: %d. Finish: %d.\n",c[0], c[N-1]);

	// release resources
    hipFree(_a);
    hipFree(_b);
    hipFree(_c);

    free(a);
    free(b);
    free(c);

    return 0;
}
