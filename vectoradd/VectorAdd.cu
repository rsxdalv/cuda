
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuAdd(int *a,int *b,int *c, int N)
{
	// global index
    int offset = blockDim.x * blockIdx.x + threadIdx.x;
    if(offset < N)
    {
        c[offset] = a[offset] + b[offset];
    }
}

#define N (1<<20)

int main()
{
	// host 
    int *a, *b, *c;

	// device
	int *_a, *_b, *_c;

    const int length = N * sizeof( int );

    hipMalloc( (void **) &_a, length );
    hipMalloc( (void **) &_b, length );
    hipMalloc( (void **) &_c, length );

    a = (int *) malloc(length);
    b = (int *) malloc(length);
    c = (int *) malloc(length);

    for(int i=0; i < N; i++)
    {
        a[i]=1;
		b[i]=2;
    }

    hipMemcpy(_a, a, length, hipMemcpyHostToDevice);
    hipMemcpy(_b, b, length, hipMemcpyHostToDevice);

    //int blocks = length/THREADS_PER_BLOCK;
	size_t blockSize = 1024; 
	size_t gridSize  = (N + blockSize - 1)/blockSize;
    cuAdd<<< gridSize, blockSize>>>(_a, _b, _c, length);

    hipMemcpy(c, _c, length, hipMemcpyDeviceToHost);

    printf("Start: %d. Finish: %d.\n",c[0], c[N-1]);

    hipFree(_a);
    hipFree(_b);
    hipFree(_c);

    free(a);
    free(b);
    free(c);


    return 0;
}
